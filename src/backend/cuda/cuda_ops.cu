#include "hip/hip_runtime.h"
/**
 * @file cuda_ops.cu
 * @brief CUDA GPU-accelerated tensor operations
 * @author Muhammad Fiaz
 * 
 * Implements GPU kernels for high-performance tensor operations using CUDA.
 * Provides parallel implementations of arithmetic operations for NVIDIA GPUs.
 */

#include "tensr/tensr.h"
#include <hip/hip_runtime.h>

/**
 * @brief CUDA kernel for element-wise addition
 * @param a First input array
 * @param b Second input array
 * @param c Output array
 * @param n Number of elements
 * 
 * GPU kernel that performs parallel element-wise addition: c[i] = a[i] + b[i]
 */
__global__ void add_kernel(float* a, float* b, float* c, size_t n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

/**
 * @brief CUDA kernel for element-wise multiplication
 * @param a First input array
 * @param b Second input array
 * @param c Output array
 * @param n Number of elements
 * 
 * GPU kernel that performs parallel element-wise multiplication: c[i] = a[i] * b[i]
 */
__global__ void mul_kernel(float* a, float* b, float* c, size_t n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] * b[idx];
    }
}

extern "C" {

/**
 * @brief Launch CUDA addition kernel
 * @param a First input array on GPU
 * @param b Second input array on GPU
 * @param c Output array on GPU
 * @param n Number of elements
 * 
 * Host function that launches the GPU addition kernel with appropriate
 * grid and block dimensions.
 */
void cuda_add(float* a, float* b, float* c, size_t n) {
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    add_kernel<<<blocks, threads>>>(a, b, c, n);
    hipDeviceSynchronize();
}

/**
 * @brief Launch CUDA multiplication kernel
 * @param a First input array on GPU
 * @param b Second input array on GPU
 * @param c Output array on GPU
 * @param n Number of elements
 * 
 * Host function that launches the GPU multiplication kernel with appropriate
 * grid and block dimensions.
 */
void cuda_mul(float* a, float* b, float* c, size_t n) {
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    mul_kernel<<<blocks, threads>>>(a, b, c, n);
    hipDeviceSynchronize();
}

}
